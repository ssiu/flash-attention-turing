#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <float.h>
#include <torch/extension.h>
#include <cute/tensor.hpp>
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>

#include "kernel_traits.h"

using namespace cute;


__global__ __launch_bounds__(64)
void compute_dq_dk_dv_kernel_v1(
    half_t const* q_ptr,
    half_t const* k_ptr,
    half_t const* v_ptr,
    float const* l_ptr,
    half_t const* do_ptr,
//     half_t* d_ptr, // dO \circ O
    half_t* dq_ptr,
    half_t* dk_ptr,
    half_t* dv_ptr,
    int batch_size, int seq_len, int num_heads, int head_dim
)
{   
    
    using MMA_Atom_Arch = MMA_Atom<SM75_16x8x8_F32F16F16F32_TN>;
    
    using TiledMma_S = TiledMMA<
        MMA_Atom_Arch,
        Layout<Shape<_2,_1,_1>>,
        Tile<_32, _32, _8>>;

    using TiledMma_dP = TiledMMA<
        MMA_Atom_Arch,
        Layout<Shape<_2,_1,_1>>,
        Tile<_32, _32, _8>>;

    using TiledMma_dV = TiledMMA<
        MMA_Atom_Arch,
        Layout<Shape<_2,_1,_1>>,
        Tile<_32, _128, _8>>;

    using SmemLayoutAtom = decltype(
                    Layout<Shape<_32, _32>,
                    Stride<_32, _1>>{});

    using SmemLayoutAtomTranposed = decltype(
                    Layout<Shape<_32, _32>,
                    Stride<_1, _32>>{});
    
    using SmemLayoutQ = decltype(
                            Layout<Shape<_32, _128>,
                            Stride<_128, _1>>{});

    using SmemLayoutQTransposed = decltype(
                                      Layout<Shape<_128, _32>,
                                      Stride<_1, _128>>{});



    using SmemLayoutKV = decltype(
           Layout<Shape<_32, _128>,
           Stride<_128, _1>>{});

    constexpr int kBlockM = 32;
    constexpr int kBlockN = 32;
    constexpr int kHeadDim = 128;
    // Q
    Tensor mQ = make_tensor(make_gmem_ptr(q_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gQ = local_tile(mQ(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockM>, Int<kHeadDim>>{},
                           make_coord(_, 0));


    // K
    Tensor mK = make_tensor(make_gmem_ptr(k_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gK = local_tile(mK(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockN>, Int<kHeadDim>>{},
                           make_coord(blockIdx.z, 0));

    // V
    Tensor mV = make_tensor(make_gmem_ptr(v_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gV = local_tile(mV(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockN>, Int<kHeadDim>>{},
                           make_coord(blockIdx.z, 0));


    // L = m + log l
    Tensor mL = make_tensor(make_gmem_ptr(l_ptr),
                             make_shape(batch_size, num_heads, seq_len),
                             make_stride(seq_len * num_heads,  seq_len, Int<1>{}));

    Tensor gL = local_tile(mL(blockIdx.x, blockIdx.y, _), Shape<Int<kBlockM>>{},
                           make_coord(_));

    // dO
    Tensor mdO = make_tensor(make_gmem_ptr(do_ptr),
                             make_shape(batch_size, seq_len, num_heads, head_dim),
                             make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gdO = local_tile(mdO(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockM>, Int<kHeadDim>>{},
                           make_coord(_, 0));

    // dV
    Tensor mdV = make_tensor(make_gmem_ptr(dv_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gdV = local_tile(mdV(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockN>, Int<kHeadDim>>{},
                           make_coord(blockIdx.z, 0));

    extern __shared__ char smem_[];

    Tensor sQ = make_tensor(make_smem_ptr(reinterpret_cast<half_t*>(&smem_[0])), SmemLayoutQ{});        // 8KB
    //Tensor sK = make_tensor(sQ.data() + kBlockM * kHeadDim, SmemLayoutKV{});
    Tensor sK = make_tensor(sQ.data() + size(sQ), SmemLayoutKV{});   // 8KB
    Tensor sV = make_tensor(sK.data() + size(sK), SmemLayoutKV{});

    Tensor sdO = make_tensor(sV.data() + size(sV), SmemLayoutQ{});                            // 8KB
    Tensor sdOt = make_tensor(sV.data() + size(sV), SmemLayoutQTransposed{});                 // 8KB

    Tensor sP = make_tensor(sdO.data() + size(sdO), SmemLayoutAtom{});                         // 2KB
    Tensor sPt = make_tensor(sdO.data() + size(sdO), SmemLayoutAtomTranposed{});               // 2KB

    Tensor sdS = make_tensor(sP.data() + size(sP), SmemLayoutAtom{});     // 2KB
    Tensor sdSt = make_tensor(sP.data() + size(sP), SmemLayoutAtomTranposed{});     // 2KB

    //Tensor sdV = make_tensor(sdS.data() + size(sdS), SmemLayoutKV{});                            // 2KB

    //int total_bytes_for_half = cosize_v<SmemLayoutQ> * 2 + cosize_v<SmemLayoutQTransposed> + cosize_v<SmemLayoutKV> * 2 + cosize_v<SmemLayoutAtom> + cosize_v<SmemLayoutAtomTranposed>;

    // only
    //Tensor sS = make_tensor(make_smem_ptr(reinterpret_cast<float*>(&smem_[0])), SmemLayoutAtom{});      // 2KB

//     if (thread0()){
//         printf("sdV size %d\n", size(sdV));
//     }

    int thread_id = threadIdx.x;
    int lane_id = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;

    int thread_row = warp_id * 16 + lane_id / 4;

    float rL[2];


    // S = QK^T
    TiledMma_S tiled_mma_S;
    ThrMMA thr_mma_S = tiled_mma_S.get_slice(threadIdx.x);
    Tensor tSgQ = thr_mma_S.partition_A(gQ);
    Tensor tSsQ = thr_mma_S.partition_A(sQ);
    Tensor tSgK = thr_mma_S.partition_B(gK);
    Tensor tSsK = thr_mma_S.partition_B(sK);
    Tensor tSrS_float = partition_fragment_C(tiled_mma_S, Shape<Int<kBlockM>, Int<kBlockN>>{});
    Tensor tSsP = thr_mma_S.partition_C(sP);
    //Tensor tSsS_float = thr_mma_S.partition_C(sS);


    // dV += P^TdO
    TiledMma_dV tiled_mma_dV;
    ThrMMA thr_mma_dV = tiled_mma_dV.get_slice(threadIdx.x);
    Tensor tdVsPt = thr_mma_dV.partition_A(sPt);
    Tensor tdVgdO = thr_mma_dV.partition_A(gdO);
    Tensor tdVsdO = thr_mma_dV.partition_A(sdO);
    Tensor tdVsdOt = thr_mma_dV.partition_B(sdOt);
    Tensor tdVrdOt = thr_mma_dV.partition_fragment_B(sdOt);

    Tensor tdVrdV_float = partition_fragment_C(tiled_mma_dV, Shape<Int<kBlockN>, Int<kHeadDim>>{});
    //Tensor tdVsdV = thr_mma_dV.partition_C(sdV);
    Tensor tdVgdV = thr_mma_dV.partition_C(gdV);

    // dP = dOV^T
    TiledMma_dP tiled_mma_dP;
    ThrMMA thr_mma_dP = tiled_mma_dP.get_slice(threadIdx.x);
    Tensor tdPgdO = thr_mma_dP.partition_A(gdO);
    Tensor tdPsdO = thr_mma_dP.partition_A(sdO);
    Tensor tdPgV = thr_mma_dP.partition_B(gV);
    Tensor tdPsV = thr_mma_dP.partition_B(sV);
    Tensor tdPrdP_float = partition_fragment_C(tiled_mma_dP, Shape<Int<kBlockM>, Int<kBlockN>>{});
    Tensor tdPsdS = thr_mma_dP.partition_C(sdS);


    auto Q_TILE_MAX = size<3>(tSgQ);

    // load K, V, dK, dV tiles
    copy(tSgK, tSsK);
    copy(tdPgV, tdPsV);

    //clear(tdVrdV_float);
    clear(tSrS_float);
    CUTE_NO_UNROLL
    for (int q_tile = 0; q_tile < Q_TILE_MAX; ++q_tile) {

        // load gQ to sQ
        copy(tSgQ(_,_,_,q_tile), tSsQ);
        copy(tdVgdO(_,_,_,q_tile), tdVsdO);


        __syncthreads();
        // compute S=QK^T
        gemm(tiled_mma_S, tSsQ, tSsK, tSrS_float);

        gemm(tiled_mma_dP, tdPsdO, tdPsV, tdPrdP_float);
        //copy(tSrS_float, tSsS_float);
        __syncthreads();


        // load rL, rD from gmem to rmem
        for (int i=0; i<2; i++) {
            rL[i] = gL((thread_row + 8 * i), q_tile);
        }


        // rescale S
        for (int i=0;i< tSrS_float.size();i ++ ) {
            tSrS_float[i] *= 1.0f / sqrtf(kHeadDim);
        }

        //copy(tSrS_float, tSsS_float);

        // compute P = exp(S-l)

        // P has size blockM x blockN, partitioned by mma_S
        // gL has size (32), need to figure the L_i for each S_ij

        for (int i=0; i<2; i++) {
            for (int j=0; j< tSrS_float(make_coord(_,i),_,_).size(); j++) {
                tSrS_float(make_coord(_,i),_,_)[j] = expf(tSrS_float(make_coord(_,i),_,_)[j] - rL[i]);
            }
        }

        // compute dS = P \circ (dP - D)
//         for (int i=0; i<2; i++) {
//             for (int j=0; j< tdPrdP_float(make_coord(_,i),_,_).size(); j++) {
//                 tdPrdP_float(make_coord(_,i),_,_)[j] = tSrS_float(make_coord(_,i),_,_)[j] * (tdPrdP_float(make_coord(_,i),_,_)[j] - rD[i]);
//             }
//         }




        //convert P from fp32 to fp16
        constexpr int num_element = decltype(size(tSrS_float))::value;

        cutlass::NumericArrayConverter<half_t, float, num_element> convert_op;
        auto frag = convert_op(*reinterpret_cast<const cutlass::Array<float, num_element> *>(tSrS_float.data()));

        Tensor tSrP = make_tensor(make_rmem_ptr<half_t>(&frag), tSrS_float.layout());
//
        copy(tSrP, tSsP);
//
        __syncthreads();

        clear(tSrS_float);

        gemm(tiled_mma_dV, tdVsPt, tdVsdOt, tdVrdV_float);

        __syncthreads();

    }

    constexpr int num_element = decltype(size(tdVrdV_float))::value;

    cutlass::NumericArrayConverter<half_t, float, num_element> convert_op;
    auto frag = convert_op(*reinterpret_cast<const cutlass::Array<float, num_element> *>(tdVrdV_float.data()));

    Tensor tdVrdV = make_tensor(make_rmem_ptr<half_t>(&frag), tdVrdV_float.layout());

//    copy(tdVrdV, tdVsdV);
    copy(tdVrdV, tdVgdV);


}



std::vector<torch::Tensor>
flash_bwd_v1(torch::Tensor q,
          torch::Tensor k,
          torch::Tensor v,
          torch::Tensor o,
          torch::Tensor l,
          torch::Tensor d_o,
          int batch_size, int seq_len, int num_heads, int head_dim)
{

    constexpr int kBlockM = 32;
    constexpr int kBlockN = 32;
    constexpr int kHeadDim = 128;

    torch::Tensor dq = torch::empty(q.sizes(), q.options().dtype(torch::kFloat16));
    torch::Tensor dk = torch::empty(k.sizes(), k.options().dtype(torch::kFloat16));
    torch::Tensor dv = torch::empty(v.sizes(), v.options().dtype(torch::kFloat16));

    half_t* q_ptr = reinterpret_cast<half_t*>(q.data_ptr());
    half_t* k_ptr = reinterpret_cast<half_t*>(k.data_ptr());
    half_t* v_ptr = reinterpret_cast<half_t*>(v.data_ptr());
    half_t* o_ptr = reinterpret_cast<half_t*>(o.data_ptr());
    float* l_ptr = reinterpret_cast<float*>(l.data_ptr());
    half_t* do_ptr = reinterpret_cast<half_t*>(d_o.data_ptr());

    half_t* dq_ptr = reinterpret_cast<half_t*>(dq.data_ptr());
    half_t* dk_ptr = reinterpret_cast<half_t*>(dk.data_ptr());
    half_t* dv_ptr = reinterpret_cast<half_t*>(dv.data_ptr());

    // compute dO \circ O
    //compute_dot_do_o


    dim3 dimGrid(batch_size, num_heads, seq_len / kBlockN);
    dim3 dimBlock(64);
    int maxbytes = 65536;


    // compute dQ, dK, dV

    hipFuncSetAttribute(reinterpret_cast<const void*>(compute_dq_dk_dv_kernel_v1), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);


   compute_dq_dk_dv_kernel_v1<<<dimGrid, dimBlock, maxbytes>>>(q_ptr,
                                            k_ptr,
                                            v_ptr,
                                            l_ptr,
                                            do_ptr,
                                            dq_ptr,
                                            dk_ptr,
                                            dv_ptr,
                                            batch_size, seq_len, num_heads, head_dim);

    return { dq, dk, dv };

}