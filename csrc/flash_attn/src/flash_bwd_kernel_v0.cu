#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <float.h>
#include <torch/extension.h>
#include <cute/tensor.hpp>
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>

#include "kernel_traits.h"

using namespace cute;


__global__ __launch_bounds__(64)
void compute_dq_dk_dv_kernel_v0(
    half_t const* q_ptr,
    half_t const* k_ptr,
    half_t const* v_ptr,
    float const* l_ptr,
    half_t const* do_ptr,
//     half_t* d_ptr, // dO \circ O
    half_t* dq_ptr,
    half_t* dk_ptr,
    half_t* dv_ptr,
    int batch_size, int seq_len, int num_heads, int head_dim
)
{   
    
    using MMA_Atom_Arch = MMA_Atom<SM75_16x8x8_F32F16F16F32_TN>;
    
    using TiledMma_S = TiledMMA<
        MMA_Atom_Arch,
        Layout<Shape<_2,_1,_1>>,
        Tile<_32, _32, _8>>;

    using TiledMma_dV = TiledMMA<
        MMA_Atom_Arch,
        Layout<Shape<_2,_1,_1>>,
        Tile<_32, _128, _8>>;

    using SmemLayoutAtom = decltype(
                    Layout<Shape<_32, _32>,
                    Stride<_32, _1>>{});

    using SmemLayoutAtomTranposed = decltype(
                    Layout<Shape<_32, _32>,
                    Stride<_1, _32>>{});
    
    using SmemLayoutQ = decltype(
                            Layout<Shape<_32, _128>,
                            Stride<_128, _1>>{});

    using SmemLayoutQTransposed = decltype(
                                      Layout<Shape<_128, _32>,
                                      Stride<_1, _128>>{});



    using SmemLayoutKV = decltype(
           Layout<Shape<_32, _128>,
           Stride<_128, _1>>{});

    constexpr int kBlockM = 32;
    constexpr int kBlockN = 32;
    constexpr int kHeadDim = 128;
    // Q
    Tensor mQ = make_tensor(make_gmem_ptr(q_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gQ = local_tile(mQ(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockM>, Int<kHeadDim>>{},
                           make_coord(_, 0));


    // K
    Tensor mK = make_tensor(make_gmem_ptr(k_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gK = local_tile(mK(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockN>, Int<kHeadDim>>{},
                           make_coord(blockIdx.z, 0));


    // L = m + log l
    Tensor mL = make_tensor(make_gmem_ptr(l_ptr),
                             make_shape(batch_size, num_heads, seq_len),
                             make_stride(seq_len * num_heads,  seq_len, Int<1>{}));

    Tensor gL = local_tile(mL(blockIdx.x, blockIdx.y, _), Shape<Int<kBlockM>>{},
                           make_coord(_));

    // dO
    Tensor mdOt = make_tensor(make_gmem_ptr(do_ptr),
                             make_shape(batch_size, head_dim, num_heads, seq_len),
                             make_stride(seq_len * num_heads * head_dim, Int<1>{}, head_dim, num_heads * head_dim));

    Tensor gdOt = local_tile(mdOt(blockIdx.x, _, blockIdx.y, _), Shape<Int<kHeadDim>, Int<kBlockM>>{},
                           make_coord(0, _));

    // dV
    Tensor mdV = make_tensor(make_gmem_ptr(dv_ptr),
                            make_shape(batch_size, seq_len, num_heads, head_dim),
                            make_stride(seq_len * num_heads * head_dim, num_heads * head_dim, head_dim, Int<1>{}));

    Tensor gdV = local_tile(mdV(blockIdx.x, _, blockIdx.y, _), Shape<Int<kBlockN>, Int<kHeadDim>>{},
                           make_coord(blockIdx.z, 0));

    extern __shared__ char smem_[];

    Tensor sQ = make_tensor(make_smem_ptr(reinterpret_cast<half_t*>(&smem_[0])), SmemLayoutQ{});
    Tensor sK = make_tensor(sQ.data() + kBlockM * kHeadDim, SmemLayoutKV{});
    Tensor sdO = make_tensor(sK.data() + kBlockN * kHeadDim, SmemLayoutQ{});
    Tensor sdOt = make_tensor(sK.data() + kBlockN * kHeadDim, SmemLayoutQTransposed{});

    Tensor sP = make_tensor(sdO.data() + kBlockM * kHeadDim, SmemLayoutAtom{});
    Tensor sPt = make_tensor(sdO.data() + kBlockM * kHeadDim, SmemLayoutAtomTranposed{});
    Tensor sdV = make_tensor(sP.data() + kBlockM * kBlockN, SmemLayoutKV{});

    int thread_id = threadIdx.x;
    int warp_id = threadIdx.x / 32;
    int thread_row = warp_id * 16 + thread_id / 4;

    float rL[2];


    // S = QK^T
    TiledMma_S tiled_mma_S;
    ThrMMA thr_mma_S = tiled_mma_S.get_slice(threadIdx.x);
    Tensor tSgQ = thr_mma_S.partition_A(gQ);
    Tensor tSsQ = thr_mma_S.partition_A(sQ);
    Tensor tSgK = thr_mma_S.partition_B(gK);
    Tensor tSsK = thr_mma_S.partition_B(sK);
    Tensor tSrS_float = partition_fragment_C(tiled_mma_S, Shape<Int<kBlockM>, Int<kBlockN>>{});
    Tensor tSsP = thr_mma_S.partition_C(sP);


    // dV += P^TdO
    TiledMma_dV tiled_mma_dV;
    ThrMMA thr_mma_dV = tiled_mma_dV.get_slice(threadIdx.x);
    Tensor tdVsPt = thr_mma_dV.partition_A(sPt);
    Tensor tdVgdOt = thr_mma_dV.partition_B(gdOt);
    Tensor tdVsdOt = thr_mma_dV.partition_B(sdOt);
    Tensor tdVrdV_float = partition_fragment_C(tiled_mma_dV, Shape<Int<kBlockN>, Int<kHeadDim>>{});
    Tensor tdVsdV = thr_mma_dV.partition_C(sdV);
    Tensor tdVgdV = thr_mma_dV.partition_C(gdV);

    auto Q_TILE_MAX = size<3>(tSgQ);

    // load K, V, dK, dV tiles
    copy(tSgK, tSsK);
    if (thread0()) {
        print(gdOt);
        print("\n");
//         for (int i =0; i<128;i++) {
//             printf("%f\n", static_cast<float>(gdOt(i,0,0)));
//         }
        for (int i =0; i< tdVgdOt.size();i++) {
            printf("%f\n", static_cast<float>(tdVgdOt[i]));
        }

    }

    clear(tdVrdV_float);

//     CUTE_NO_UNROLL
//     for (int q_tile = 0; q_tile < Q_TILE_MAX; ++q_tile) {
//         clear(tSrS_float);
//
// //         for (int i=0;i < tSrS_float.size();i ++ ) {
// //             tSrS_float[i] = 0;
// //             if (thread0()) {
// //                 printf("reset tSrS\n");
// //                 printf("%f ", tSrS_float[i]);
// //                 printf("\n");
// //             }
// //         }
//
//
//
//         // load gQ to sQ
//         copy(tSgQ(_,_,_, q_tile), tSsQ);
//         //copy(tdVgdOt(_,_,_, q_tile), tdVsdOt);
//
//         __syncthreads();
//         // compute S=QK^T
//         gemm(tiled_mma_S, tSsQ, tSsK, tSrS_float);
//
//
//         // load rL, rD from gmem to rmem
//         for (int i=0; i<2; i++) {
//             rL[i] = gL((thread_row + 8 * i), q_tile);
//         }
//
//         if (thread0()) {
//             printf("tSrS\n");
//             for (int i =0;i < tSrS_float.size(); i++){
//                 printf("%f ", tSrS_float[i]);
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//         // rescale S
//         for (int i=0;i< tSrS_float.size();i ++ ) {
//             tSrS_float[i] *= 1.0f / sqrtf(kHeadDim);
//         }
//
//         if (thread0()) {
//             printf("tSrS after scaling headdim\n");
//             for (int i =0;i < tSrS_float.size(); i++){
//                 printf("%f ", tSrS_float[i]);
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//         // compute P = exp(S-l)
//
//         // P has size blockM x blockN, partitioned by mma_S
//         // gL has size (32), need to figure the L_i for each S_ij
//
//         for (int i=0; i<2; i++) {
//             for (int j=0; j< tSrS_float(make_coord(_,i),_,_).size(); j++) {
//                 tSrS_float(make_coord(_,i),_,_)[j] = expf(tSrS_float(make_coord(_,i),_,_)[j] - rL[i]);
//             }
//         }
//
//         if (thread0()) {
//             printf("tSrP float\n");
//             for (int i =0;i < tSrS_float.size(); i++){
//                 printf("%f ", tSrS_float[i]);
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//         //convert P from fp32 to fp16
//         constexpr int num_element = decltype(size(tSrS_float))::value;
//
//         cutlass::NumericArrayConverter<half_t, float, num_element> convert_op;
//         auto frag = convert_op(*reinterpret_cast<const cutlass::Array<float, num_element> *>(tSrS_float.data()));
//
//         Tensor tSrP = make_tensor(make_rmem_ptr<half_t>(&frag), tSrS_float.layout());
//
//         if (thread0()) {
//             printf("tSrP\n");
//             for (int i =0;i < tSrS_float.size(); i++){
//                 printf("%f ", static_cast<float>(tSrS_float[i]));
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//         if (thread0()) {
//
//             printf("tdVgdOt\n");
//             print(gdOt);
//             print(tdVgdOt);
//             for (int i =0;i < 10; i++){
// //                 printf("%f ", static_cast<float>(tdVgdOt[i]));
//                 printf("%f ", static_cast<float>(gdOt[i]));
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//         if (thread0()) {
//             printf("tdVsdOt\n");
//             print(sdOt);
//             print(tdVsdOt);
// //             for (int i =0;i < tdVsdOt.size(); i++){
// //                 printf("%f ", tdVsdOt[i]);
// //             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//
// //
//         copy(tSrP, tSsP);
// //
//         __syncthreads();
//         gemm(tiled_mma_dV, tdVsPt, tdVsdOt, tdVrdV_float);
//
//         __syncthreads();
//
//         if (thread0()) {
//             printf("tdVrdV\n");
//             for (int i =0;i < tdVrdV_float.size(); i++){
//                 printf("%f ", tdVrdV_float[i]);
//             }
//             print("\n");
//             print("=====");
//             print("\n");
//         }
//
//
//
//     }
//
//
//     if (thread0()) {
//         printf("tdVrdV, FINISH\n");
//         for (int i =0;i < tdVrdV_float.size(); i++){
//             printf("%f ", tdVrdV_float[i]);
//         }
//         print("\n");
//
//     }
//
//     constexpr int num_element = decltype(size(tdVrdV_float))::value;
//
//     cutlass::NumericArrayConverter<half_t, float, num_element> convert_op;
//     auto frag = convert_op(*reinterpret_cast<const cutlass::Array<float, num_element> *>(tdVrdV_float.data()));
//
//     Tensor tdVrdV = make_tensor(make_rmem_ptr<half_t>(&frag), tdVrdV_float.layout());
//
// //    copy(tdVrdV, tdVsdV);
//     copy(tdVrdV, tdVgdV);
//
// //     dq_ptr[0] = static_cast<half_t>(0.0f);
// //     dk_ptr[0] = static_cast<half_t>(0.0f);
// //     dv_ptr[0] = static_cast<half_t>(0.0f);
//
//     if (thread0()) {
//         print(gQ);
//         print("\n");
//         print(sQ);
//         print("\n");
//         print(tSgQ);
//         print("\n");
//         print(tSsQ);
//         print("\n");
//         print(gK);
//         print("\n");
//         print(sK);
//         print("\n");
//         print(tSgK);
//         print("\n");
//         print(tSsK);
//         print("\n");
//         print(sP);
//         print("\n");
//         print(sPt);
//         print("\n");
//         print(gdOt);
//         print("\n");
//         print(sdOt);
//         print("\n");
//         print(gdV);
//         print("\n");
//         print(sdV);
//         print("\n");
//         print(tdVrdV);
//         print("\n");
//         print(tdVsdV);
//         print("\n");
//         print(tdVgdV);
//         print("\n");
//         print(gL);
//         print("\n");
//     }
}



std::vector<torch::Tensor>
flash_bwd_v0(torch::Tensor q,
          torch::Tensor k,
          torch::Tensor v,
          torch::Tensor o,
          torch::Tensor l,
          torch::Tensor d_o,
          int batch_size, int seq_len, int num_heads, int head_dim)
{

    constexpr int kBlockM = 32;
    constexpr int kBlockN = 32;
    constexpr int kHeadDim = 128;

    torch::Tensor dq = torch::empty(q.sizes(), q.options().dtype(torch::kFloat16));
    torch::Tensor dk = torch::empty(k.sizes(), k.options().dtype(torch::kFloat16));
    torch::Tensor dv = torch::empty(v.sizes(), v.options().dtype(torch::kFloat16));

    half_t* q_ptr = reinterpret_cast<half_t*>(q.data_ptr());
    half_t* k_ptr = reinterpret_cast<half_t*>(k.data_ptr());
    half_t* v_ptr = reinterpret_cast<half_t*>(v.data_ptr());
    half_t* o_ptr = reinterpret_cast<half_t*>(o.data_ptr());
    float* l_ptr = reinterpret_cast<float*>(l.data_ptr());
    half_t* do_ptr = reinterpret_cast<half_t*>(d_o.data_ptr());

    half_t* dq_ptr = reinterpret_cast<half_t*>(dq.data_ptr());
    half_t* dk_ptr = reinterpret_cast<half_t*>(dk.data_ptr());
    half_t* dv_ptr = reinterpret_cast<half_t*>(dv.data_ptr());

    // compute dO \circ O
    //compute_dot_do_o


    dim3 dimGrid(batch_size, num_heads, seq_len / kBlockN);
    dim3 dimBlock(64);
    int maxbytes = 65536;


    // compute dQ, dK, dV

    hipFuncSetAttribute(reinterpret_cast<const void*>(compute_dq_dk_dv_kernel_v0), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);


   compute_dq_dk_dv_kernel_v0<<<dimGrid, dimBlock, maxbytes>>>(q_ptr,
                                            k_ptr,
                                            v_ptr,
                                            l_ptr,
                                            do_ptr,
                                            dq_ptr,
                                            dk_ptr,
                                            dv_ptr,
                                            batch_size, seq_len, num_heads, head_dim);

    return { dq, dk, dv };

}